#include "hip/hip_runtime.h"

#include "cuda_config.hpp"
#include "routines/add_source.hpp"

__global__ void add_source_kernel(float_type *p, float_type src, size_t ix, size_t iz)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i > 0)
        return;
    p[i * iz + ix] = src;
}

template <>
void add_source(ScalarField<TMP::MemSpaceCuda> &p, const float_type src, size_t ix, size_t iz, TMP::ExecutionSpaceCuda)
{
    assert(ix < p.get_nx());
    assert(iz < p.get_nz());
#if defined(TMP_ENABLE_CUDA_BACKEND)
    add_source_kernel<<<1, 1>>>(p.get_ptr(), src, ix, iz);
#elif defined(TMP_ENABLE_HIP_BACKEND)
    static_assert(false, "NOT IMPLEMENTED YET");
#endif
}
